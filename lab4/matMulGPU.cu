#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <random>
#include <chrono>
#include <string>
#include <sstream>
#include <iomanip>
#include "hip/hip_runtime.h"

using namespace std;

// Error checking macro for CUDA calls
#define CHECK_CUDA_ERROR(call) \
{ \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        cerr << "CUDA error in " << #call << ": " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << endl; \
        exit(EXIT_FAILURE); \
    } \
}

// Kernel for matrix multiplication
__global__ void matrixMultiplyKernel(const int* A, const int* B, int* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int get_random_number(int min, int max) {
    static random_device rd;
    static mt19937 gen(rd());
    uniform_int_distribution<int> dist(min, max);
    return dist(gen);
}

vector<vector<int>> generate_matrix(int size, int minVal = 1, int maxVal = 100) {
    vector<vector<int>> matrix(size, vector<int>(size));
    for (auto& row : matrix) {
        for (auto& elem : row) {
            elem = get_random_number(minVal, maxVal);
        }
    }
    return matrix;
}

void save_matrix(const vector<vector<int>>& matrix, const string& filename) {
    ofstream out(filename);
    for (const auto& row : matrix) {
        for (size_t i = 0; i < row.size(); ++i) {
            out << row[i];
            if (i != row.size() - 1) out << " ";
        }
        out << "\n";
    }
}

vector<vector<int>> read_matrix(const string& filename) {
    ifstream in(filename);
    vector<vector<int>> matrix;
    string line;

    while (getline(in, line)) {
        vector<int> row;
        istringstream iss(line);
        int num;
        while (iss >> num) {
            row.push_back(num);
        }
        matrix.push_back(row);
    }

    return matrix;
}

vector<vector<int>> multiply_matrices_cuda(const vector<vector<int>>& A, const vector<vector<int>>& B) {
    int n = A.size();

    // Flatten matrices
    vector<int> A_flat(n * n);
    vector<int> B_flat(n * n);
    vector<int> C_flat(n * n);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            A_flat[i * n + j] = A[i][j];
            B_flat[i * n + j] = B[i][j];
        }
    }

    // Allocate device memory
    int* d_A, * d_B, * d_C;
    CHECK_CUDA_ERROR(hipMalloc(&d_A, n * n * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, n * n * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_C, n * n * sizeof(int)));

    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpy(d_A, A_flat.data(), n * n * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, B_flat.data(), n * n * sizeof(int), hipMemcpyHostToDevice));

    // Define block and grid dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    matrixMultiplyKernel << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, n);
    CHECK_CUDA_ERROR(hipGetLastError());

    // Copy result back to host
    CHECK_CUDA_ERROR(hipMemcpy(C_flat.data(), d_C, n * n * sizeof(int), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_C));

    // Convert flat result back to 2D vector
    vector<vector<int>> result(n, vector<int>(n));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            result[i][j] = C_flat[i * n + j];
        }
    }

    return result;
}

int main() {
    // Check for CUDA device
    int deviceCount;
    CHECK_CUDA_ERROR(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        cerr << "No CUDA devices found" << endl;
        return EXIT_FAILURE;
    }
    cout << "Found " << deviceCount << " CUDA device(s)" << endl;

    const string results_dir = "E://PPfix/Paral_1/lab4/results";
    ofstream stats("statistics_cuda.txt");

    for (int size = 100; size <= 4100; size += 500) {
        cout << "Size: " << size << "x" << size << endl;

        auto matrix1 = generate_matrix(size);
        auto matrix2 = generate_matrix(size);

        string file1 = results_dir + "/" + to_string(size) + "_1.txt";
        string file2 = results_dir + "/" + to_string(size) + "_2.txt";
        save_matrix(matrix1, file1);
        save_matrix(matrix2, file2);

        auto start = chrono::high_resolution_clock::now();
        auto result = multiply_matrices_cuda(matrix1, matrix2);
        auto end = chrono::high_resolution_clock::now();

        string result_file = results_dir + "/result_cuda_" + to_string(size) + ".txt";
        save_matrix(result, result_file);

        auto duration = chrono::duration_cast<chrono::milliseconds>(end - start);
        stats << size << "\t" << duration.count() << " ms\n";

        cout << "  CUDA time: " << duration.count() << " ms\n";

        
    }
    stats.close();
    cout << "Check statistics_cuda.txt for results\n";
    return 0;
}